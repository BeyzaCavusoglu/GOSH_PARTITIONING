#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_functions.h"
// fastSig()
#include "sigmoid.h"
// randn()
#include "rand_helper.h"
#include <stdio.h>
#include <float.h>

using namespace std;

#define MAX_ALPHA 101
#define FULL_MASK 0xffffffff
__inline__ __device__ 
double warpAllReduceSum(double val) {

  for (int i =1; i<32; i*=2){
    val+=__shfl_xor_sync(FULL_MASK,val,i);
  }
  return val;
}

__device__ vid_t get_negative_sample(vid_t num_vertices, void* seed){
  return randn(seed) % num_vertices;
}
/////// For testing ///////
__device__ bool u_has_edge_v(vid_t u, vid_t v, vid_t* d_V, vid_t* d_A){
  for (unsigned long long i =0; i< d_V[u+1]-d_V[u];i++){
    if (d_A[i+d_V[u]]==v) return true;
  }
  return false;
}

__device__ bool u_in_part_x(vid_t u, int part_id, int vids_per_bin, int num_vertices){
  vid_t min = vids_per_bin*part_id;
  vid_t max = vids_per_bin*(part_id+1);
  if (max>num_vertices) max = num_vertices;
  if (u>=max || u<min) return false;
  else return true;
}
//////////////////////////

//*********************************************************
// BLOOM FILTER

__device__ bool testBit2(BF filter, int bit){
  
  int index = bit / 64;
  int offset = bit % 64;
  
  return (filter.filter[index] & (1ULL << offset)) != 0;
  
}


__device__ void printFilter2(BF filter){

  printf("##Filter: ");
  
  for(int i = 0; i < FILTER_SIZE; i++){ //Traverse bits
    if(testBit2(filter, i))
      printf("1");
    else
      printf("0");
  }
  printf("\n");
}

__device__ int popcllAll2(BF filter){

  int total_flipped = 0;

  for(int i = 0; i < FILTER_SIZE / 64; i++){ //traverses "index"
    total_flipped += __popcll(filter.filter[i]);
  }

  return total_flipped;
} 

__device__ int findCommon2(BF filter, BF filter2, int source, int p_sample){


  int common_flipped = 0;


    for(int i = 0; i < FILTER_SIZE; i++){ //Traverse bits

    if(testBit2(filter, i) && testBit2(filter2,i))
    {
      common_flipped += 1;
    } 
     
}

  return common_flipped/2;
}


__device__ int calculate_bandwidth(unsigned int * vertexList, unsigned int * edgeList){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int bandwidth = 0;
  if (tid < sizeof(vertexList)) 
  {
    int x = vertexList[tid];
    int x_next = vertexList[tid + 1]; 
    int num_of_neighbors = x_next - x;

    for (int j= x ; j < x + num_of_neighbors; j++)
    {
        int neighbor_vertex = edgeList[j];
        bandwidth = max(bandwidth, abs(tid - neighbor_vertex));
    }
 
  }
  return bandwidth;
}
 

// BLOOM FILTER ENDS
//**********************************************************
__device__ double sum_f;
__device__ double min_f = DBL_MAX;
__device__ double max_f = -DBL_MAX;
__device__ int num_f_values = 0;

__device__ void single_sample_update(emb_t * vEmbeddings, emb_t * uEmbeddings, int dimensions, int positive, double learning_rate, int id, double bias, float* d_sigmoid_lookup_table, double negative_weight, int WARP_SIZE, unsigned int source, unsigned int generated_sample, unsigned int* bloomFilter, unsigned int * vertexList, int bf, int common, int total_source, int total_psample, unsigned int* edgeList, double* d_f_values){
  int start = id % WARP_SIZE;
  double fake_jaccard = (double)common/(total_source + total_psample - common); 
  double myscore = 0;
  for (int i=start ; i < dimensions; i += WARP_SIZE){
    myscore += vEmbeddings[i] * uEmbeddings[i];
  }
  double g = warpAllReduceSum(myscore);
  g-=bias;
  double f=0, f2 =0 ;
  if(bf == 1){
    sum_f += fake_jaccard ;
    num_f_values++;
    double avg_f = sum_f / num_f_values;
    f = (positive - fastSig(g, d_sigmoid_lookup_table)* (fake_jaccard/avg_f)) * learning_rate ; 
      d_f_values[id] = f;
     }
  else{//bloom filter off, so no fake jaccard, only original gosh calculation
    f = (positive - fastSig(g, d_sigmoid_lookup_table)) * learning_rate;
      d_f_values[id] = f; 
       }
  // perform update on embedding of graphs
  double nw = 1;
  if (positive == 0){ nw = negative_weight;  }
  for (int i = start; i < dimensions; i += WARP_SIZE){
    float u = uEmbeddings[i];
    float v = vEmbeddings[i];
    vEmbeddings[i] += u * f * nw;
    uEmbeddings[i] += v * f * nw;
  }
}


//wedge parametresi eklendi, exec komutunda 1 ya da 0 verilmesine gore wedge sampling yapcak
__device__ unsigned int get_positive_sample_ppr_device(unsigned int source, unsigned int * V, unsigned int * A, void* seed, int alpha, int wedge){
  unsigned int result = source;
  unsigned int numNeighbours = V[result+1] - V[result];
  unsigned long randNum = randn(seed);
  if (alpha == 0){
    if (numNeighbours == 0){
      return result;
    }
     else {
      if (wedge == 0){
        return A[V[result] + (randNum%numNeighbours)];
      }
      else{ //wedge sampling aciksa
      // KOMSUSU YERINE KOMSUSUNUN KOMSUSUNU DONCEM WEDGE ICIN |WEDGE PATH|=2
        unsigned int w = A[V[result] + (randNum%numNeighbours)];
        unsigned int numNeighboursNeighbours = V[w+1] - V[w];
        unsigned long randNumNum = randn(seed);


        return A[V[w] + (randNumNum%numNeighboursNeighbours)];
      }
    }
  }
  while(randNum % MAX_ALPHA < alpha){
    if (numNeighbours == 0){
      return result;
    }
    else {
      result = A[V[result]+(randNum%numNeighbours)];
    }
    randNum = randn(seed);
  }
  return result;
} 




__global__ void Embedding_Kernel(unsigned int d_num_vertices, int d_num_epoch, unsigned int * d_V, unsigned int * d_A,  emb_t * d_embeddings, int d_dim, int d_s, double d_lr, float* d_sigmoid_lookup_table, int ep_start, int total_batches, int alpha, int wedge, int bf, double negative_weight, unsigned int* bloomFilter, BF* d_filters, double* d_f_values, int WARP_SIZE, int WARPS_PER_BLOCK, int NUM_WARPS){

  
  //A warp per vertex strategy
  const unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;

  
  const unsigned int warp_num = id/WARP_SIZE;
  const double nce_bias = logf(d_num_vertices);
  const double nce_neg_bias=logf(d_num_vertices/(d_s));
  double lr = d_lr;
  //seed for random number generation
  seed mySeed;
  void *sd = &mySeed; // This is an abstraction to allow us to pass any seed to the functions (for other random function tests later)
  mySeed.x = (1+warp_num*ep_start)*123456789; // the values of the seed are a function of the thread id
  mySeed.y = (1+warp_num*ep_start)*362436069; // (i wasn't very creative with these functions but I didn't see any decrease in ML performance)
  mySeed.z = (1+warp_num*ep_start)*521288629;
  //
  unsigned int source, p_sample, n_sample;
  // Every block has 128*number of warps within it shared memory floats
  // each warp will store its embeddings somewhere in this big array
  // myemb_s size = 128*(NUM_THREADS/WARP_SIZE)
  // warp_id_in_block = warp_num%WARPS_PER_BLOCK
  const int dims = d_dim;
  extern __shared__ emb_t emb_s[];
  emb_t* myemb_s = &emb_s[(warp_num%WARPS_PER_BLOCK)*dims];
  emb_t* myemb_g;
  for (int ep = 0; ep < d_num_epoch; ep++){
    for (int i = warp_num; i< d_num_vertices; i+=NUM_WARPS){
      source = i;
      sum_f = 0;
      p_sample = get_positive_sample_ppr_device(source, d_V, d_A, sd, alpha, wedge);//generate positive sample
      if(p_sample != UINT_MAX) {
        myemb_g = d_embeddings+(dims*source);
        for (int j =id%WARP_SIZE; j<dims; j+=WARP_SIZE)
          myemb_s[j] = myemb_g[j];

        // CAN COMPUTE FAKE JACCARD HERE, GIVE IT TO SINGLE SAMPLE UPDATE
        int common = findCommon2(d_filters[source], d_filters[p_sample], source, p_sample);

       // printf("source:%d | sample:%d | common:%d\n", source, p_sample, common);
        int total_source = popcllAll2(d_filters[source]);
        int total_psample= popcllAll2(d_filters[p_sample]);

	      //Will interfere from here
        single_sample_update(myemb_s, d_embeddings+dims*p_sample, dims, 1, lr, id, nce_bias, d_sigmoid_lookup_table, negative_weight, WARP_SIZE, source, p_sample, bloomFilter, d_V, bf, common, total_source, total_psample, d_A, d_f_values); //update one positive sample
        // generated sample: either p_sample or n_sample will be given inside.
        
        for(unsigned int k = 0; k < d_s; k++){
          n_sample = get_negative_sample(d_num_vertices, sd);//get negative sample
          single_sample_update(myemb_s, d_embeddings+dims*n_sample, dims, 0, lr, id, nce_neg_bias, d_sigmoid_lookup_table, negative_weight, WARP_SIZE, source, n_sample, bloomFilter, d_V, bf, common, total_source, total_psample, d_A, d_f_values);
        }
        for (int l =id%WARP_SIZE; l<dims; l+=WARP_SIZE)
          myemb_g[l] = myemb_s[l];
      }
    }
  }
}


__global__ void Big_Graphs_Embedding_Kernel(emb_t *source_bin, emb_t* dest_bin, long long vertices_per_part, int num_vertices, int starting_ep, int batch_ep,vid_t* vids, double d_lr, int dim, int neg_s, float* sig_table, int alpha, int wedge, int bf, int source_part_id, int dest_part_id , int WARP_SIZE, int WARPS_PER_BLOCK, int NUM_WARPS,  unsigned int* bloomFilter, BF* d_filters, unsigned int* vertexList, unsigned int *edgeList){
 
  int common2 =0;
  int total_source2 = 0;
  int total_psample2 = 0;
  double* d_f_values;
  const unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
  const unsigned int warp_num = id/WARP_SIZE;
  const double nce_bias = logf(1.0*num_vertices);
  const double nce_neg_bias=logf(1.0*num_vertices/float(neg_s));
  emb_t* myemb_g;
  long long size_s2d=vids[0];
  long long size_d2s = vids[1];
  vids = &vids[2];
  long long w;
#ifdef SHARED_MEMORY
  extern __shared__ emb_t emb_s[];
  emb_t* myemb_s = &emb_s[(warp_num%WARPS_PER_BLOCK)*dim];
#else
  emb_t*& myemb_s = myemb_g;
#endif
  seed mySeed;
  mySeed.x = (1+warp_num+starting_ep)*123456789; // the values of the seed are a function of the thread id
  mySeed.y = (1+warp_num+starting_ep)*362436069; // (i wasn't very creative with these functions but I didn't see any decrease in ML performance)
  mySeed.z = (1+warp_num+starting_ep)*521288629;
  vid_t source, p_sample, ns;
  double lr = d_lr;
  vid_t ns_limit = vertices_per_part;
  if ((dest_part_id+1)*vertices_per_part>num_vertices){
    ns_limit = num_vertices-vertices_per_part*dest_part_id;
  }
  for (w = warp_num; w<size_s2d; w+=NUM_WARPS){
    // p_sample is the absolute id of the target
    source = vids[w*2];
#ifdef _DEBUG_GPU
    if (!u_in_part_x(source, source_part_id, vertices_per_part, num_vertices))
      printf("PROBLEM: s2d - Found a positive sample while processing %d, %d that is incorrect\n", source_part_id, dest_part_id);
#endif
    myemb_g = source_bin+(dim*(source-vertices_per_part*source_part_id));
#ifdef SHARED_MEMORY
    for (int j =id%WARP_SIZE; j<dim; j+=WARP_SIZE)
      myemb_s[j] = myemb_g[j];
#endif
    p_sample =vids[w*2+1];
#ifdef _DEBUG_GPU
    if (!u_in_part_x(p_sample, dest_part_id, vertices_per_part, num_vertices))
      printf("PROBLEM: s2d - Found a positive sample while processing %d, %d that is incorrect\n", source_part_id, dest_part_id);
#endif
    single_sample_update(myemb_s, dest_bin+(p_sample-vertices_per_part*dest_part_id)*dim, dim, 1, lr, id, nce_bias, sig_table, 1, WARP_SIZE, source, p_sample, bloomFilter, vertexList, bf, common2,  total_source2, total_psample2, edgeList,  d_f_values);
    for (int s = 0;s <neg_s; s++){
      ns = get_negative_sample(ns_limit, &mySeed);
      single_sample_update(myemb_s, dest_bin+ns*dim, dim, 0, lr, id, nce_neg_bias, sig_table, 1, WARP_SIZE, source, ns, bloomFilter, vertexList, bf, common2, total_source2, total_psample2, edgeList, d_f_values);
    }
#ifdef SHARED_MEMORY
    for (int j =id%WARP_SIZE; j<dim; j+=WARP_SIZE)
      myemb_g[j] = myemb_s[j];
#endif
  }
  if (source_part_id!=dest_part_id){
    ns_limit = vertices_per_part;
    if ((source_part_id+1)*vertices_per_part>num_vertices){
      ns_limit = num_vertices-vertices_per_part*source_part_id;
    }
    long long offset = 2*(size_s2d);
    for (w = offset+(warp_num*2); w<offset+(size_d2s*2); w+=2*NUM_WARPS){
      // p_sample is the absolute id of the target
      source = vids[(w)];
#ifdef _DEBUG_GPU
      if (!u_in_part_x(source, dest_part_id, vertices_per_part, num_vertices))
        printf("PROBLEM: d2s - Found a sourcewhile processing %d, %d that is incorrect\n", source_part_id, dest_part_id);
#endif
      myemb_g = dest_bin+(dim*(source-vertices_per_part*dest_part_id));
#ifdef SHARED_MEMORY
      for (int j =id%WARP_SIZE; j<dim; j+=WARP_SIZE)
        myemb_s[j] = myemb_g[j];
#endif
      p_sample =vids[w+1];
#ifdef _DEBUG_GPU
      if (!u_in_part_x(p_sample,source_part_id, vertices_per_part, num_vertices))
        printf("PROBLEM: d2s - Found a positive sample while processing %d, %d that is incorrect\n", source_part_id, dest_part_id);
#endif
      single_sample_update(myemb_s, source_bin+(p_sample-vertices_per_part*source_part_id)*dim, dim, 1, lr, id, nce_bias, sig_table, 1, WARP_SIZE, source, p_sample, bloomFilter, vertexList, bf, common2, total_source2, total_psample2, edgeList, d_f_values);
      for (int s = 0;s <neg_s; s++){
        ns = get_negative_sample(ns_limit, &mySeed);
        single_sample_update(myemb_s, source_bin+ns*dim, dim, 0, lr, id, nce_neg_bias, sig_table, 1, WARP_SIZE, source, ns, bloomFilter, vertexList, bf, common2, total_source2, total_psample2, edgeList, d_f_values);
      }
#ifdef SHARED_MEMORY
      for (int j =id%WARP_SIZE; j<dim; j+=WARP_SIZE)
        myemb_g[j] = myemb_s[j];
#endif
    }
  }  
  //if (id == 0) printf("finished %d %d\n",source_part_id, dest_part_id); 
}
const unsigned kFullMask = 0xFFFFFFFF;
  template <class T>
    __device__ T WarpBroadcast(T value, int lane_id) {
#if __CUDACC_VER_MAJOR__ >= 9
      return __shfl_sync(kFullMask, value, lane_id);
#else
      return __shfl(value, lane_id);
#endif
    }
  template <class T>
    __device__ T WarpReduce(T value) {
#pragma unroll
      for (int delta = 1; delta < 32; delta *= 2)
#if __CUDACC_VER_MAJOR__ >= 9
        value += __shfl_down_sync(kFullMask, value, delta);
#else
      value += __shfl_down(value, delta);
#endif
      return value;
    }

#define MAX_ALPHA 101
#define FULL_MASK 0xffffffff
#define NEGATIVE_WEIGHT 5
__device__ void single_sample_update_pos(emb_t * vEmbeddings, emb_t * uEmbeddings, float learning_rate, int id, float bias, int dimension, int WARP_SIZE){
   int start = id % WARP_SIZE;
   //double myscore = 0;
   float x = 0;
   for (int i=start ; i < dimension; i += WARP_SIZE){
     x += vEmbeddings[i] * uEmbeddings[i];
   }
   x = WarpBroadcast(WarpReduce(x), 0);
   x-=bias;
   float prob = x > 0 ? 1 / (1 + exp(-x)) : exp(x) / (exp(x) + 1);
   prob = prob - 1;
   // perform update on embedding of graphs
   float lol = prob  * learning_rate;
   for (int i = start; i < dimension; i += WARP_SIZE){
     float u = uEmbeddings[i];
     float v = vEmbeddings[i];
     vEmbeddings[i] -= u * lol;
     uEmbeddings[i] -= v * lol;
   }
}
__device__ void single_sample_update_neg(emb_t * vEmbeddings, emb_t * uEmbeddings, float learning_rate, int id, float bias, int dimension, float negative_weight, int WARP_SIZE){
   int start = id % WARP_SIZE;
   //double myscore = 0;
   float x = 0;
   for (int i=start ; i < dimension; i += WARP_SIZE){
     x += vEmbeddings[i] * uEmbeddings[i];
   }
   x = WarpBroadcast(WarpReduce(x), 0);
   x-=bias;
   float prob = x > 0 ? 1 / (1 + exp(-x)) : exp(x) / (exp(x) + 1);
   // perform update on embedding of graphs
   float lol = prob *negative_weight * learning_rate;
   for (int i = start; i < dimension; i += WARP_SIZE){
     float u = uEmbeddings[i];
     float v = vEmbeddings[i];
     vEmbeddings[i] -= u * lol;
     uEmbeddings[i] -= v * lol;
   }
}
__global__ void Embedding_Kernel_SP(unsigned int d_num_vertices, unsigned long samples_per_pool, unsigned int* d_sample_array, unsigned int* d_fake,  emb_t * d_embeddings, float d_lr, int dimension, int negative_samples, float negative_weight, int WARPS_PER_BLOCK, int WARP_SIZE, int NUM_WARPS){
  //A warp per vertex strategy
  const unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
  const unsigned int warp_num = id/WARP_SIZE;
  const float nce_bias = logf(d_num_vertices);
  const float nce_neg_bias=logf(d_num_vertices/(negative_samples));
  unsigned int source, p_sample, n_sample;
  extern __shared__ emb_t emb_s[];
  emb_t* myemb_s = &emb_s[(warp_num%WARPS_PER_BLOCK)*dimension]; 
  emb_t* myemb_g;
  for (int i = warp_num; i<samples_per_pool; i+=NUM_WARPS){
    source = d_sample_array[i*2];
    p_sample = d_sample_array[i*2+1];
    myemb_g = d_embeddings+(dimension*source);
    for (int j =id%WARP_SIZE; j<dimension; j+=WARP_SIZE)
      myemb_s[j] = myemb_g[j];
    single_sample_update_pos(myemb_s, d_embeddings+dimension*p_sample, d_lr, id, nce_bias, dimension, WARP_SIZE); //update one positive sample
    for(unsigned int k = 0; k < negative_samples; k++){
      n_sample = d_fake[i*negative_samples+k];
      single_sample_update_neg(myemb_s, d_embeddings+dimension*n_sample, d_lr, id, nce_neg_bias, dimension, negative_weight, WARP_SIZE);
    }
    for (int l =id%WARP_SIZE; l<dimension; l+=WARP_SIZE)
      myemb_g[l] = myemb_s[l];
  }
}

